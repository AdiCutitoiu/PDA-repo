#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <cstdlib>

__global__ void RoyFloydStep(int k, int * m, int noNodes)
{
  int& thisElem = m[threadIdx.x * noNodes + threadIdx.y];

  int otherFirstElem  = m[threadIdx.x * noNodes + k];
  int otherSecondElem = m[k * noNodes + threadIdx.y];

  if(thisElem > otherFirstElem + otherSecondElem)
    thisElem = otherFirstElem + otherSecondElem;
}

int main()
{
  int m[][5] = {
    { 0, 3, 9, 8, 3 },
    { 5, 0, 1, 4, 2 },
    { 6, 6, 0, 4, 5 },
    { 2, 9, 2, 0, 7 },
    { 7, 9, 3, 2, 0 }
  };

  int * matrix;
  hipMalloc(&matrix, _countof(m) * _countof(m) * sizeof(int));
  hipMemcpy(matrix, m, _countof(m) * _countof(m) * sizeof(int), hipMemcpyHostToDevice);

  dim3 threadsPerBlock(_countof(m), _countof(m));
  for (int k = 0; k < _countof(m); k++)
  {
    RoyFloydStep<<<1, threadsPerBlock>>>(k, matrix, _countof(m));
  }

  hipMemcpy(m, matrix, _countof(m) * _countof(m) * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(matrix);

  for (const auto & line : m)
  {
    for (auto elem : line)
    {
      std::cout << elem << ' ';
    }
    std::cout << '\n';
  }

  return 0;
}
