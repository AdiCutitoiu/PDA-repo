#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <cstdlib>

__global__ void RoyFloydStep(int k, int * m, int noNodes)
{
  int& thisElem = m[threadIdx.x * noNodes + threadIdx.y];

  int otherFirstElem  = m[threadIdx.x * noNodes + k];
  int otherSecondElem = m[k * noNodes + threadIdx.y];

  if(thisElem > otherFirstElem + otherSecondElem)
    thisElem = otherFirstElem + otherSecondElem;
}

int main()
{
  int h_m[][5] = {
    { 0, 3, 9, 8, 3 },
    { 5, 0, 1, 4, 2 },
    { 6, 6, 0, 4, 5 },
    { 2, 9, 2, 0, 7 },
    { 7, 9, 3, 2, 0 }
  };

  int * d_matrix;
  hipMalloc(&d_matrix, _countof(h_m) * _countof(h_m) * sizeof(int));
  hipMemcpy(d_matrix, h_m, _countof(h_m) * _countof(h_m) * sizeof(int), hipMemcpyHostToDevice);

  dim3 threadsPerBlock(_countof(h_m), _countof(h_m));
  for (int k = 0; k < _countof(h_m); k++)
  {
    RoyFloydStep<<<1, threadsPerBlock>>>(k, d_matrix, _countof(h_m));
  }

  hipMemcpy(h_m, d_matrix, _countof(h_m) * _countof(h_m) * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_matrix);

  for (const auto & line : h_m)
  {
    for (auto elem : line)
    {
      std::cout << elem << ' ';
    }
    std::cout << '\n';
  }

  return 0;
}
